#include "hip/hip_runtime.h"
//   Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "paddle/fluid/framework/trie_manager.h"
#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace paddle {
namespace framework {
const int CUDA_NUM_THREADS = platform::PADDLE_CUDA_NUM_THREADS;
#define GET_BLOCK(N) ((N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS)

#define CUDA_KERNEL_LOOP(i, n)                                  \
  for (auto i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)
#define CUDA_BLOCK(N) GET_BLOCK(N), CUDA_NUM_THREADS, 0

__global__ void fill_mask_kernel(const int len, const int N, const int64_t* pos, int64_t* arr) {
  CUDA_KERNEL_LOOP(i, len) {
    int x = pos[i*2];
    int y = pos[i*2+1];

    arr[x * N + y] = 1;
  }
}

}  // end namespace framework
}  // end namespace paddle